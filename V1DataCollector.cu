
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
// here you can put any values you want for n
// warning do not change the length of the array
int nValues[15]={100,150,200,250,350,500,650,800,900,1000,1200,1400,1600,1800,2000};
// here you can put any values you want for k
// warning do not change the length of the array
int kValues[5]={10,20,45,80,100};
void initializeG(int n, int *G){
  for (int i=0;i<n*n;i++){
    if((random()%2)==0)
    G[i]=1;
    else
    G[i]=-1;
}
  }
  __device__
  static void calculate(int *readingArray, int* writingArray, double *weights, int n ,int current,int xAxes, int yAxes){
  		double Sum = 0;
  		if(current < n*n)
  		{
  	// loop through all the points that affect
  	for(int p=-2;p<3;p++){
  	for(int q=-2;q<3;q++){
  	Sum += weights[(p+2)*5+(q+2)] * readingArray[((p + yAxes + n) % n) * n + ( q + xAxes + n) % n];
  	// index properly in order to include the wrap points
  	// add the weight to Sum
  	}
  	}

  	// check to decide which value the current spin should take
  			if(Sum > 0.00001)// set to 0.000001 in order to take into account
  			// floating points
  				writingArray[current] = 1;
  			else if(Sum < -0.00001)
  				writingArray[current] = -1;
  			else // if it is zero then let the value remain the same
  				writingArray[current] = readingArray[current];
  		}
  }
  // cuda function to parallelize the spin calculation
  __global__ void spinCalculation(int n, double * gpuWeights,int *gpuG,int *gpuGTemp,int i,int block) {
   // variable to hold the sum of the weights

  	int current = blockIdx.x * block * block + threadIdx.x; // calculation of the current index



  	int xAxes = current % n; // calculate x axes
  	int yAxes = current / n; // calculate y axes




  	// switch the i%2 which is the current number of iretarion
  	// so periodically we will be writing to gpuGTemp and then to gpuG
  switch (i%2) {
  	case 0:
  	 calculate(gpuG,gpuGTemp,gpuWeights,n,current,xAxes,yAxes);
  	break;
  // here everything is the same with the difference that is reading from the gpuGTemp array
  // and write to the gpuG
  case 1:
  calculate(gpuGTemp,gpuG,gpuWeights,n,current,xAxes,yAxes);
  }
  }
  void ising (int *G, double *w, int k, int n,int grid ,int block)
  {

  double *weights; // declare double pointer to pass weights to gpu
   hipMalloc(&weights,sizeof(double)*25); // allocate memoery

  hipMemcpy(weights,w,25*sizeof(double),hipMemcpyHostToDevice);// copy
  //data from host to device

  int *tempG=(int *) malloc(sizeof(int)*n*n); //allocate memory for tempG

  memcpy(tempG,G,n*n*sizeof(int)); // coppy G to temp G
    int *gpuTempG; // pointer to pass to device representing tempG
  	hipMalloc(&gpuTempG,n*n*sizeof(int)); // allocate memory
    int *gpuG; // pointer to pass to device representing G
  	hipMalloc(&gpuG,n*n*sizeof(int)); //allocate device memory for gpuG
    hipMemcpy(gpuTempG,tempG,n*n*sizeof(int),hipMemcpyHostToDevice);
  	//copy tempG to device memory
    hipMemcpy(gpuG,G,n*n*sizeof(int),hipMemcpyHostToDevice);
  	//copy G to device memory
  //loop k times
  for(int i=0;i<k;i++){
  	spinCalculation<<<grid*grid,block*block>>>(n,weights,gpuG,gpuTempG,i,block);//
  	// launch kernel function to execute in parallel
    hipDeviceSynchronize(); // sunchronize
  }
  // again if k is odd the datas are in gpuTempG and if it is even in gpuG
  if(k%2==1){
  hipMemcpy(G,gpuTempG,n*n*sizeof(int),hipMemcpyDeviceToHost);
  }
  else{
  	hipMemcpy(G,gpuG,n*n*sizeof(int),hipMemcpyDeviceToHost);
  }

  // free memory
  hipFree(gpuG);
  hipFree(gpuTempG);
  free(tempG);
  }

int main(){
int block=50;
int grid;
FILE *file; // pointer to a file
clock_t end,start; // variables to count time
int n,k; // n and k
  double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004,
  			0.016, 0.071, 0.117, 0.071, 0.016,
  		0.026, 0.117, 0, 0.117, 0.026,
  		0.016, 0.071, 0.117, 0.071, 0.016,
  		0.004, 0.016, 0.026, 0.016, 0.004}; // array with weights
// loop through every n and k value
for(int i=0;i<15;i++){
n=nValues[i]; // set n value
grid=n/block +1;
int *G=(int*)malloc(sizeof(int)*n*n); // allocate memory for G
for(int j=0;j<5;j++){


k=kValues[j]; // set k value
initializeG(n,G);// initialize G
start=clock(); // start counting
ising(G,weights,k,n,grid,block);// call ising
end=clock();// stop counting
file=fopen("V1.csv","a"); // open csv file
fprintf(file, "%d ,%d, %lf\n",n,k, ((double)(end-start))/CLOCKS_PER_SEC);
// write the data
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC);// print data
fclose(file);// close file
}
free(G);// free G
}
	return 0;
}
