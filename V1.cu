
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
static int grid_array[5]={50,60,70,80,90};
static int block_array[5]={11,15,20,25,30};
__device__
static void calculate(int *readingArray, int* writingArray, double *weights, int n ,int current,int xAxes, int yAxes){
		double Sum = 0;
		if(current < n*n)
		{
	// loop through all the points that affect
	for(int p=-2;p<3;p++){
	for(int q=-2;q<3;q++){
	Sum += weights[(p+2)*5+(q+2)] * readingArray[((p + yAxes + n) % n) * n + ( q + xAxes + n) % n];
	// index properly in order to include the wrap points
	// add the weight to Sum
	}
	}

	// check to decide which value the current spin should take
			if(Sum > 0.00001)// set to 0.000001 in order to take into account
			// floating points
				writingArray[current] = 1;
			else if(Sum < -0.00001)
				writingArray[current] = -1;
			else // if it is zero then let the value remain the same
				writingArray[current] = readingArray[current];
		}
}
// cuda function to parallelize the spin calculation
__global__ void spinCalculation(int n, double * gpuWeights,int *gpuG,int *gpuGTemp,int i,int block) {
 // variable to hold the sum of the weights

	int current = blockIdx.x * block * block + threadIdx.x; // calculation of the current index



	int xAxes = current % n; // calculate x axes
	int yAxes = current / n; // calculate y axes




	// switch the i%2 which is the current number of iretarion
	// so periodically we will be writing to gpuGTemp and then to gpuG
switch (i%2) {
	case 0:
	 calculate(gpuG,gpuGTemp,gpuWeights,n,current,xAxes,yAxes);
	break;
// here everything is the same with the difference that is reading from the gpuGTemp array
// and write to the gpuG
case 1:
calculate(gpuGTemp,gpuG,gpuWeights,n,current,xAxes,yAxes);
}
}
void takeBinData(int *array, FILE *file,int n){

if (file==NULL)
{
printf("error opening file");
exit(1);
}
fread(array,sizeof(int),n*n,file);
fclose(file);
}
void ising (int *G, double *w, int k, int n,int grid ,int block)
{

double *weights; // declare double pointer to pass weights to gpu
 hipMalloc(&weights,sizeof(double)*25); // allocate memoery

hipMemcpy(weights,w,25*sizeof(double),hipMemcpyHostToDevice);// copy
//data from host to device

int *tempG=(int *) malloc(sizeof(int)*n*n); //allocate memory for tempG

memcpy(tempG,G,n*n*sizeof(int)); // coppy G to temp G
  int *gpuTempG; // pointer to pass to device representing tempG
	hipMalloc(&gpuTempG,n*n*sizeof(int)); // allocate memory
  int *gpuG; // pointer to pass to device representing G
	hipMalloc(&gpuG,n*n*sizeof(int)); //allocate device memory for gpuG
  hipMemcpy(gpuTempG,tempG,n*n*sizeof(int),hipMemcpyHostToDevice);
	//copy tempG to device memory
  hipMemcpy(gpuG,G,n*n*sizeof(int),hipMemcpyHostToDevice);
	//copy G to device memory
//loop k times
for(int i=0;i<k;i++){
	spinCalculation<<<grid*grid,block*block>>>(n,weights,gpuG,gpuTempG,i,block);//
	// launch kernel function to execute in parallel
  hipDeviceSynchronize(); // sunchronize
}
// again if k is odd the datas are in gpuTempG and if it is even in gpuG
if(k%2==1){
hipMemcpy(G,gpuTempG,n*n*sizeof(int),hipMemcpyDeviceToHost);
}
else{
	hipMemcpy(G,gpuG,n*n*sizeof(int),hipMemcpyDeviceToHost);
}

// free memory
hipFree(gpuG);
hipFree(gpuTempG);
free(tempG);
}

void checkCorrectness(int *G, int *expectedState,int n,int k){
bool noMistake=true;
int counter=0;
	for(int i=0;i<n*n;i++)
	{
		if(expectedState[i]!=G[i])
			{
				//printf("wrong in index %d\n",i );
counter++;
	noMistake=false;

	}
	}
	if (noMistake) {
	printf("ising for k=%d is correct\n",k );
	}
	else{
		printf("ising for k=%d is wrong\n",k );

	}
printf("%d\n",counter );
}

int main(){
int n=517;
int grid,block;
for(int i=0;i<5;i++){
	for(int j=0;j<5;j++)
{
grid=grid_array[i];
block=block_array[j];
int *initialG=(int *) malloc(sizeof(int)*n*n);
int *G=(int *)malloc(sizeof(int)*n*n);
int *expectedState=(int *)malloc(sizeof(int)*n*n);
FILE *file;
file= fopen("conf-init.bin","rb");
takeBinData(initialG,file,n);
memcpy(G,initialG,sizeof(int)*n*n);
double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004,
			0.016, 0.071, 0.117, 0.071, 0.016,
		0.026, 0.117, 0, 0.117, 0.026,
		0.016, 0.071, 0.117, 0.071, 0.016,
		0.004, 0.016, 0.026, 0.016, 0.004};
clock_t start,end;
start=clock();
ising(G,weights,1,n,grid,block);
end=clock();
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC);
file=fopen("conf-1.bin","rb");
takeBinData(expectedState,file,n);
checkCorrectness(G,expectedState,n,1);
memcpy(G,initialG,sizeof(int)*n*n);
start=clock();
ising(G,weights,4,n,grid,block);
end=clock();
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC);
file =fopen("conf-4.bin","rb");
takeBinData(expectedState,file,n);
checkCorrectness(G,expectedState,n,4);
memcpy(G,initialG,sizeof(int)*n*n);
start=clock();
ising(G,weights,11,n,grid,block);
	end=clock();
	printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC);
file =fopen("conf-11.bin","rb");
takeBinData(expectedState,file,n);
checkCorrectness(G,expectedState,n,11);
}
}

	return 0;
}
