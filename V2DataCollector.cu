
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdbool.h>
int nValues[15]={100,150,200,250,350,500,650,800,900,1000,1200,1400,1600,1800,2000};
// here you can put any values you want for k
// warning do not change the length of the array
int kValues[5]={10,20,45,80,100};
__device__
static void calculate(int *readingArray, int* writingArray, double *weights, int n ,int current,int xAxes, int yAxes){
		double Sum = 0;
		if(current < n*n)
		{
	// loop through all the points that affect
	for(int p=-2;p<3;p++){
	for(int q=-2;q<3;q++){
	Sum += weights[(p+2)*5+(q+2)] * readingArray[((p + yAxes + n) % n) * n + ( q + xAxes + n) % n];
	// index properly in order to include the wrap points
	// add the weight to Sum
	}
	}

	// check to decide which value the current spin should take
			if(Sum > 0.00001)// set to 0.000001 in order to take into account
			// floating points
				writingArray[current] = 1;
			else if(Sum < -0.00001)
				writingArray[current] = -1;
			else // if it is zero then let the value remain the same
				writingArray[current] = readingArray[current];
		}
}
// cuda function to parallelize the spin calculation
__global__ void spinCalculation(int n, double * gpuWeights,int *gpuG,int *gpuGTemp,int i,int block,int looper) {
 // variable to hold the sum of the weights

	int current = blockIdx.x * block * block + threadIdx.x; // calculation of the current index



	int xAxes;
	int yAxes;


for(int q=0;q<looper;q++)
{
	// switch the i%2 which is the current number of iretarion
	// so periodically we will be writing to gpuGTemp and then to gpuG
switch (i%2) {
	case 0:
xAxes=(current*looper+q)%n;
yAxes=(current*looper+q)/n;
	 calculate(gpuG,gpuGTemp,gpuWeights,n,current*looper+q,xAxes,yAxes);
	break;
// here everything is the same with the difference that is reading from the gpuGTemp array
// and write to the gpuG
case 1:
xAxes=(current*looper+q)%n;
yAxes=(current*looper+q)/n;
calculate(gpuGTemp,gpuG,gpuWeights,n,current*looper+q,xAxes,yAxes);
break;
}
}
}
void ising (int *G, double *w, int k, int n,int grid ,int block)
{
int looper= n*n/(grid*grid*block*block) + 1;
double *weights;
 hipMalloc(&weights,sizeof(double)*25);

hipMemcpy(weights,w,25*sizeof(double),hipMemcpyHostToDevice);

int *tempG=(int *) malloc(sizeof(int)*n*n);

memcpy(tempG,G,n*n*sizeof(int));
  int *gpuTempG;
	hipMalloc(&gpuTempG,n*n*sizeof(int));
  int *gpuG;
	hipMalloc(&gpuG,n*n*sizeof(int));
  hipMemcpy(gpuTempG,tempG,n*n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gpuG,G,n*n*sizeof(int),hipMemcpyHostToDevice);

for(int i=0;i<k;i++){
	spinCalculation<<<grid*grid,block*block>>>(n,weights,gpuG,gpuTempG,i,block,looper);
  hipDeviceSynchronize();
}
if(k%2==1){
hipMemcpy(G,gpuTempG,n*n*sizeof(int),hipMemcpyDeviceToHost);
}
else{
	hipMemcpy(G,gpuG,n*n*sizeof(int),hipMemcpyDeviceToHost);
}


hipFree(gpuG);
hipFree(gpuTempG);
free(tempG);
}
void initializeG(int n, int *G){
  for (int i=0;i<n*n;i++){
    if((random()%2)==0)
    G[i]=1;
    else
    G[i]=-1;
}
  }
int main(){
int block=50;
int grid=50;
FILE *file; // pointer to a file
clock_t end,start; // variables to count time
int n,k; // n and k
  double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004,
  			0.016, 0.071, 0.117, 0.071, 0.016,
  		0.026, 0.117, 0, 0.117, 0.026,
  		0.016, 0.071, 0.117, 0.071, 0.016,
  		0.004, 0.016, 0.026, 0.016, 0.004}; // array with weights
// loop through every n and k value
for(int i=0;i<15;i++){
n=nValues[i]; // set n value

int *G=(int*)malloc(sizeof(int)*n*n); // allocate memory for G
for(int j=0;j<5;j++){


k=kValues[j]; // set k value
initializeG(n,G);// initialize G
start=clock(); // start counting
ising(G,weights,k,n,grid,block);// call ising
end=clock();// stop counting
file=fopen("V2.csv","a"); // open csv file
fprintf(file, "%d ,%d, %lf\n",n,k, ((double)(end-start))/CLOCKS_PER_SEC);
// write the data
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC);// print data
fclose(file);// close file
}
free(G);// free G
}
	return 0;
}
