
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdbool.h>
static int grid_array[5]={5,9,16,23,30};
static int block_array[5]={2,3,5,10,12};
static FILE *pointerToFile;
__device__
static void calculate(int *readingArray, int* writingArray, double *weights, int n ,int current,int xAxes, int yAxes){
		double Sum = 0;
		if(current < n*n)
		{
	// loop through all the points that affect
	for(int p=-2;p<3;p++){
	for(int q=-2;q<3;q++){
	Sum += weights[(p+2)*5+(q+2)] * readingArray[((p + yAxes + n) % n) * n + ( q + xAxes + n) % n];
	// index properly in order to include the wrap points
	// add the weight to Sum
	}
	}

	// check to decide which value the current spin should take
			if(Sum > 0.00001)// set to 0.000001 in order to take into account
			// floating points
				writingArray[current] = 1;
			else if(Sum < -0.00001)
				writingArray[current] = -1;
			else // if it is zero then let the value remain the same
				writingArray[current] = readingArray[current];
		}
}
// cuda function to parallelize the spin calculation
__global__ void spinCalculation(int n, double * gpuWeights,int *gpuG,int *gpuGTemp,int i,int block,int looper) {
 // variable to hold the sum of the weights

	int current = blockIdx.x * block * block + threadIdx.x; // calculation of the current index



	int xAxes;
	int yAxes;


for(int q=0;q<looper;q++)
{
	// switch the i%2 which is the current number of iretarion
	// so periodically we will be writing to gpuGTemp and then to gpuG
switch (i%2) {
	case 0:
xAxes=(current*looper+q)%n;
yAxes=(current*looper+q)/n;
	 calculate(gpuG,gpuGTemp,gpuWeights,n,current*looper+q,xAxes,yAxes);
	break;
// here everything is the same with the difference that is reading from the gpuGTemp array
// and write to the gpuG
case 1:
xAxes=(current*looper+q)%n;
yAxes=(current*looper+q)/n;
calculate(gpuGTemp,gpuG,gpuWeights,n,current*looper+q,xAxes,yAxes);
break;
}
}
}
void takeBinData(int *array, FILE *file,int n){

if (file==NULL)
{
printf("error opening file");
exit(1);
}
fread(array,sizeof(int),n*n,file);
fclose(file);
}
void ising (int *G, double *w, int k, int n,int grid ,int block)
{
int looper= n*n/(grid*grid*block*block) + 1;
double *weights;
 hipMalloc(&weights,sizeof(double)*25);

hipMemcpy(weights,w,25*sizeof(double),hipMemcpyHostToDevice);

int *tempG=(int *) malloc(sizeof(int)*n*n);

memcpy(tempG,G,n*n*sizeof(int));
  int *gpuTempG;
	hipMalloc(&gpuTempG,n*n*sizeof(int));
  int *gpuG;
	hipMalloc(&gpuG,n*n*sizeof(int));
  hipMemcpy(gpuTempG,tempG,n*n*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gpuG,G,n*n*sizeof(int),hipMemcpyHostToDevice);

for(int i=0;i<k;i++){
	spinCalculation<<<grid*grid,block*block>>>(n,weights,gpuG,gpuTempG,i,block,looper);
  hipDeviceSynchronize();
}
if(k%2==1){
hipMemcpy(G,gpuTempG,n*n*sizeof(int),hipMemcpyDeviceToHost);
}
else{
	hipMemcpy(G,gpuG,n*n*sizeof(int),hipMemcpyDeviceToHost);
}


hipFree(gpuG);
hipFree(gpuTempG);
free(tempG);
}

void checkCorrectness(int *G, int *expectedState,int n,int k){
bool noMistake=true;
int counter=0;
	for(int i=0;i<n*n;i++)
	{
		if(expectedState[i]!=G[i])
			{
				//printf("wrong in index %d\n",i );
counter++;
	noMistake=false;

	}
	}
	if (noMistake) {
	printf("ising for k=%d is correct\n",k );
	}
	else{
		printf("ising for k=%d is wrong\n",k );

	}
printf("%d\n",counter );
}

int main(){
int n=517;
int grid,block;
for(int i=0;i<5;i++){
	for(int j=0;j<5;j++)
{
grid=grid_array[i];
block=block_array[j];
int *initialG=(int *) malloc(sizeof(int)*n*n);
int *G=(int *)malloc(sizeof(int)*n*n);
int *expectedState=(int *)malloc(sizeof(int)*n*n);
FILE *file;
file= fopen("conf-init.bin","rb");
takeBinData(initialG,file,n);
memcpy(G,initialG,sizeof(int)*n*n);
double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004,
			0.016, 0.071, 0.117, 0.071, 0.016,
		0.026, 0.117, 0, 0.117, 0.026,
		0.016, 0.071, 0.117, 0.071, 0.016,
		0.004, 0.016, 0.026, 0.016, 0.004};
clock_t start,end;
start=clock();
ising(G,weights,1,n,grid,block);
end=clock();
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC);
file=fopen("conf-1.bin","rb");
takeBinData(expectedState,file,n);
checkCorrectness(G,expectedState,n,1);
memcpy(G,initialG,sizeof(int)*n*n);
start=clock();
ising(G,weights,4,n,grid,block);
end=clock();
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC );
file =fopen("conf-4.bin","rb");
takeBinData(expectedState,file,n);
checkCorrectness(G,expectedState,n,4);
memcpy(G,initialG,sizeof(int)*n*n);
start=clock();
ising(G,weights,11,n,grid,block);
end=clock();
printf("%lf\n",((double)(end-start))/CLOCKS_PER_SEC );
file =fopen("conf-11.bin","rb");
takeBinData(expectedState,file,n);
checkCorrectness(G,expectedState,n,11);
}
}

	return 0;
}
